#include "hip/hip_runtime.h"
// Multiply the arrays A' and A on GPU and save the result in C
// C(M,M) = A'(M,N) * A(N,M)
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <ctime>
#include <string>
#include <cmath>
#include <stdio.h>
#include <sys/time.h>
using namespace std;

#define cudaCheckError() {                               \
                hipError_t e = hipGetLastError();                    \
                if (e != hipSuccess) {                                \
                        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, \
                               hipGetErrorString(e));                      \
                        exit(1);                                             \
                }                                                      \
}


double *h_A, *h_C;

// Fill  random numbers on GPU
double fill_rand() {
        double s = 0, d = 0;
        //Gives a number between -20 and +20;
        s = rand() % 40 - 20;
        //Reduces this number
        d = s / 137;
        return d;
}

__global__ void multiplication(double *d_out, double *d_in, int c, int r){

        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int l;
        double myP = 0;
        if(idx <= c * r ) {
                for(l = 0; l < r; l++) {
                        myP += d_in[idx/c + l * c] * d_in[idx % c + c * l];
                }
                d_out[idx] = myP;
        }
}

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const double * A, int nr_rows_A, int nr_cols_A) {
        for (int i = 0; i < nr_rows_A; ++i) {
                for (int j = 0; j < nr_cols_A; ++j) {
                        cout << std::setprecision(15) << A[j * nr_rows_A + i] << " ";
                }
                cout << " ; "<< endl;
        }
}

int main(int argc, char * * argv) {
        srand(time(0));
        hipSetDevice(0); cudaCheckError();

        unsigned int nr_rows_A, nr_cols_A, nr_rows_C, nr_cols_C;
        if (argc < 5) {
                cout << "Usage: "<< argv[0] << " Rows Cols Loops Print" << endl;
                return 1;
        }
        if ((atoi(argv[1]) < 1) || (atoi(argv[2]) < 1)) {
                cout << "Only positive" << endl;
                return 1;
        }
        nr_rows_A = atoi(argv[1]);
        nr_cols_A = nr_rows_C = nr_cols_C = atoi(argv[2]);

        unsigned short loops = atoi(argv[3]);
        bool print = atoi(argv[4]);

        hipEvent_t start, stop;

        // Allocate 2 arrays on CPU
        double * h_A = (double * ) malloc(nr_rows_A * nr_cols_A * sizeof(double));
        double * h_C = (double * ) malloc(nr_rows_C * nr_cols_C * sizeof(double));

        // here we fill the initial array with random double numbers
        for (int i = 0; i < nr_rows_A * nr_cols_A; i++) {
                h_A[i] = fill_rand();
        }

        if (print) {
                cout << "A = [" << endl;
                print_matrix(h_A, nr_rows_A, nr_cols_A);
                cout << "];" << endl;
        }

        // Allocate 2 arrays on GPU
        double *d_A, *d_C;
        hipMalloc( &d_A, nr_rows_A * nr_cols_A * sizeof(double) ); cudaCheckError();
        hipMalloc( &d_C, nr_rows_C * nr_cols_C * sizeof(double) ); cudaCheckError();
        // Transfer the array to GPU
        hipMemcpy(d_A, h_A, nr_rows_A * nr_cols_A  * sizeof(double), hipMemcpyHostToDevice);

        cudaCheckError();
        // Αρχικοποίηση του πίνακα h_C με μηδενικά
        for (int i = 0; i < nr_rows_C * nr_cols_C; i++) {
                h_C[i] = 0.0;
        }
        hipMemcpy(d_C, h_C, nr_rows_C * nr_cols_C * sizeof(double), hipMemcpyHostToDevice);
        cudaCheckError();
        dim3 dimGrid(nr_rows_C, nr_cols_C, 1);
        hipEventCreate( &start);  cudaCheckError();
        hipEventCreate( &stop);  cudaCheckError();

        float totalTime[loops];
        int blockN = (nr_cols_A * nr_cols_A) / 1024 + 1;
        // Multiply A' and A on GPU
        for (int k = 0; k < loops; k++) {

                hipEventRecord(start);  cudaCheckError();
                // Do the actual multiplication
                multiplication<<<blockN,1024>>>(d_C, d_A, nr_cols_A, nr_rows_A);
                // multiply<<<dimGrid, 1>>>( d_A, nr_rows_A, nr_cols_A, d_C );
                hipDeviceSynchronize(); cudaCheckError();
                hipEventRecord(stop); cudaCheckError();

                // Copy (and print) the result on host memory
                hipMemcpy(h_C, d_C, nr_rows_C * nr_cols_C * sizeof(double), hipMemcpyDeviceToHost);
                cudaCheckError();
                hipEventSynchronize(stop); cudaCheckError();
                float milliseconds = 0;
                hipEventElapsedTime( &milliseconds, start, stop);
                cudaCheckError();
                cout << milliseconds <<endl;
                // cout << "\% ex2, Dimensions :" << nr_rows_A<< "x" << nr_cols_C << " , Time (msec): " << milliseconds << endl;
        }
        cout << endl;

        float mo = 0.0;
        for(int k = 0; k < loops; k++) {
                mo += totalTime[k];
        }
        mo /= (loops);
        cout << endl << "average time = " << mo << endl;

        hipEventDestroy(start); cudaCheckError();
        hipEventDestroy(stop); cudaCheckError();

        //Free GPU memory
        hipFree(d_A); cudaCheckError();
        hipFree(d_C); cudaCheckError();

        if (print) {
                cout << "C = [" << endl;
                print_matrix(h_C, nr_rows_C, nr_cols_C);
                cout << "];" << endl;
                //Για επαλήθευση σε MATLAB
                cout<< "B=A'*A; D=B-C;" << endl;
        }

        // Free CPU memory
        free(h_A);
        free(h_C);

        return 0;
}
