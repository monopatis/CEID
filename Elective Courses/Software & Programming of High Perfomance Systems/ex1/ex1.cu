// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// C(M,M) = A'(M,N) * A(N,M)

#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <string>
#include <cmath>
#include <stdio.h>
#include <sys/time.h>
using namespace std;

#define cudaCheckError() {                               \
                hipError_t e = hipGetLastError();                    \
                if (e != hipSuccess) {                                \
                        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, \
                               hipGetErrorString(e));                      \
                        exit(1);                                             \
                }                                                      \
}

double * h_A, * h_C;

// Fill  random numbers on GPU
double fill_rand() {
        double s = 0, d = 0;
        //Gives a number between -20 and +20;
        s = rand() % 40 - 20;
        //Reduces this number
        d = s / 137;
        return d;
}

// Multiply the arrays A' and A on GPU and save the result in C
// C(k,k) = A'(k,m) * A(m,k)

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const double * A, int nr_rows_A, int nr_cols_A) {
        for (int i = 0; i < nr_rows_A; ++i) {
                for (int j = 0; j < nr_cols_A; ++j) {
                        cout << std::setprecision(15) << A[j * nr_rows_A + i] << " ";
                }
                cout << " ; "<< endl;
        }
}

int main(int argc, char * * argv) {
        srand(time(0));
        hipSetDevice(0); cudaCheckError();
        if (argc < 5) {
                cout << "Usage: "<< argv[0] << " Rows Cols Loops Print" << endl;
                return 1;
        }
        unsigned int nr_rows_A, nr_cols_A, nr_rows_C, nr_cols_C;
        nr_rows_A = atoi(argv[1]);
        nr_cols_A = nr_rows_C = nr_cols_C = atoi(argv[2]);

        unsigned short loops = atoi(argv[3]);
        bool print = atoi(argv[4]);

        hipEvent_t start, stop;
        // Allocate 2 arrays on CPU
        double * h_A = (double * ) malloc(nr_rows_A * nr_cols_A * sizeof(double));
        double * h_C = (double * ) malloc(nr_rows_C * nr_cols_C * sizeof(double));

        // here we fill the initial array with random double numbers
        for (int i = 0; i < nr_rows_A * nr_cols_A; i++) {
                h_A[i] = fill_rand();
        }

        if (print) {
                cout << "A = [" << endl;
                print_matrix(h_A, nr_rows_A, nr_cols_A);
                cout << "]" << endl;
        }

        // Allocate 2 arrays on GPU
        double * d_A, * d_C;
        hipMalloc( &d_A, nr_rows_A * nr_cols_A * sizeof(double));  cudaCheckError();
        hipMalloc( &d_C, nr_rows_C * nr_cols_C * sizeof(double));  cudaCheckError();

        // Transfer the array to GPU
        hipMemcpy(d_A, h_A, nr_rows_A * nr_cols_A  * sizeof(double), hipMemcpyHostToDevice);
        cudaCheckError();
        hipEventCreate( &start);  cudaCheckError();
        hipEventCreate( &stop);  cudaCheckError();
        float totalTime[loops + 1];

        for (unsigned short k = 0; k < loops; k++) {
                const double alf = 1;
                const double bet = 0;
                const double * alpha = &alf;
                const double * beta = &bet;
                // Create a handle for CUBLAS
                hipblasHandle_t handle;  cudaCheckError();
                hipblasCreate( &handle); cudaCheckError();
                hipEventRecord(start); cudaCheckError();

                // Do the actual multiplication
                hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, nr_cols_A, nr_cols_A, nr_rows_A,
                            alpha, d_A, nr_rows_A, d_A, nr_rows_A, beta, d_C, nr_cols_A);
                cudaCheckError();

                hipEventRecord(stop); cudaCheckError();
                // Destroy the handle
                hipblasDestroy(handle); cudaCheckError();
                // Copy the result on host memory
                hipMemcpy(h_C, d_C, nr_rows_C * nr_cols_C * sizeof(double), hipMemcpyDeviceToHost); cudaCheckError();
                hipEventSynchronize(stop); cudaCheckError();
                float milliseconds = 0;
                hipEventElapsedTime( &milliseconds, start, stop); cudaCheckError();
                cout << milliseconds << endl;
                totalTime[k] = milliseconds;
                // cout << "ex1, Dimensions :" << nr_rows_A<< "x" << nr_cols_C << " , Time (msec) : " << milliseconds << endl;
        }

        hipEventDestroy(start); cudaCheckError();
        hipEventDestroy(stop); cudaCheckError();
        float mo = 0.0;
        for(int k = 0; k < loops; k++) {
                mo += totalTime[k];
        }
        mo /= (loops);
        cout << endl << "average time = " << mo << endl;

        //Free GPU memory
        hipFree(d_A); cudaCheckError();
        hipFree(d_C); cudaCheckError();

        if (print) {
                cout << "C =" << endl;
                print_matrix(h_C, nr_rows_C, nr_cols_C);
        }

        // Free CPU memory
        free(h_A);
        free(h_C);

        return 0;
}
