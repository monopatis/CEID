#include "hip/hip_runtime.h"
// Multiply the arrays A' and A on GPU and save the result in C
// C(M,M) = A'(M,N) * A(N,M)
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <ctime>
#include <string>
#include <cmath>
#include <stdio.h>
#include <sys/time.h>
using namespace std;

#define cudaCheckError() {                               \
                hipError_t e = hipGetLastError();                    \
                if (e != hipSuccess) {                                \
                        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, \
                               hipGetErrorString(e));                      \
                        exit(1);                                             \
                }                                                      \
}

#define TILE_WIDTH 8
//επιστρέφει τον μεγαλύτερο μεταξύ δύο αριθμών
#define imax(a,b) (a>b ? a : b)

double *h_A, *h_C;

// Fill  random numbers on GPU
double fill_rand() {
        double s = 0, d = 0;
        //Gives a number between -20 and +20;
        s = rand() % 40 - 20;
        //Reduces this number
        d = s / 137;
        return d;
}

//Kernel για πίνακες που δεν είναι τετραγωνικοί
/*__global__ void multiplyNdiffM( double *A, int N, int M, double *C ) {

        unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
        unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
        double Cvalue = 0.0;
        for (unsigned int e = 0; e < N; ++e)
                Cvalue += A[row * N + e] * A[col * N + e];
        C[col * M + row] = Cvalue;

   }*/

__global__ void multiplication(double *d_out, double *d_in, int c, int r){

        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int l;
        double myP = 0;
        if(idx <= c * r ) {
                for(l = 0; l < r; l++) {

                        myP += d_in[idx/c * l + c] * d_in[idx % c * c + l];

                }
                d_out[idx] = myP;
        }


}
//Kernel για τετραγωνικούς πίνακες
//Δυστυχώς δεν καταφέραμε να τον κανουμε να δουλέψει για όλους τους συνδιασμούς
//διαστάσεων. Για Ν=1 ή Μ=1 υπολογίζει σωστά τις τιμές
__global__ void multiply( double *A, int N, int M, double *C ) {
        // για να αποφύγουμε Shared Memory Bank Conflicts προσθέτουμε μία στήλη ακόμη
        __shared__ double ds_M[TILE_WIDTH][TILE_WIDTH+1];
        __shared__ double ds_N[TILE_WIDTH][TILE_WIDTH+1];
        int bx = blockIdx.x, by = blockIdx.y,
            tx = threadIdx.x, ty = threadIdx.y,
            Row = by * TILE_WIDTH + ty,
            Col = bx * TILE_WIDTH + tx;
        double Cvalue = 0.0;
        if (Row-8<Col) {
                for (int m = 0; m < (imax(N,M)-1)/TILE_WIDTH+1; ++m) {
                        if (Row < M && m*TILE_WIDTH+tx < N)
                                ds_M[ty][tx] = A[Row*N + m*TILE_WIDTH+tx];
                        else
                                ds_M[ty][tx] = 0.0;
                        if (Col < M && m*TILE_WIDTH+ty < N)
                                ds_N[ty][tx] = A[(m*TILE_WIDTH+ty)+(M*Col)];
                        else
                                ds_N[ty][tx] = 0.0;
                        __syncthreads();
                        // για TILE_WIDTH = 8 ξετύλιγμα βρόχου
                        Cvalue += ds_M[ty][0] * ds_N[0][tx]+
                                  ds_M[ty][1] * ds_N[1][tx]+
                                  ds_M[ty][2] * ds_N[2][tx]+
                                  ds_M[ty][3] * ds_N[3][tx]+
                                  ds_M[ty][4] * ds_N[4][tx]+
                                  ds_M[ty][5] * ds_N[5][tx]+
                                  ds_M[ty][6] * ds_N[6][tx]+
                                  ds_M[ty][7] * ds_N[7][tx];
                        __syncthreads();
                }
                if (Row < M && Col < M ) {
                        C[Col*M+Row] = Cvalue;
                        //ταυτόχρονα αποθηκεύουμε και το συμμετρικό του στοιχείο
                        if (Row < Col)
                                C[Row*M+Col] = Cvalue;
                }

        }
}

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const double * A, int nr_rows_A, int nr_cols_A) {
        for (int i = 0; i < nr_rows_A; ++i) {
                for (int j = 0; j < nr_cols_A; ++j) {
                        cout << std::setprecision(15) << A[j * nr_rows_A + i] << " ";
                }
                cout << " ; "<< endl;
        }
}

int main(int argc, char * * argv) {
        srand(time(0));
        hipSetDevice(0); cudaCheckError();

        unsigned int nr_rows_A, nr_cols_A, nr_rows_C, nr_cols_C;
        if (argc < 5) {
                cout << "Usage: "<< argv[0] << " Rows Cols Loops Print" << endl;
                return 1;
        }
        if ((atoi(argv[1]) < 1) || (atoi(argv[2]) < 1)) {
                cout << "Only positive" << endl;
                return 1;
        }
        nr_rows_A = atoi(argv[1]);
        nr_cols_A = nr_rows_C = nr_cols_C = atoi(argv[2]);

        unsigned short loops = atoi(argv[3]);
        bool print = atoi(argv[4]);

        hipEvent_t start, stop;

        // Allocate 2 arrays on CPU
        double * h_A = (double * ) malloc(nr_rows_A * nr_cols_A * sizeof(double));
        double * h_C = (double * ) malloc(nr_rows_C * nr_cols_C * sizeof(double));

        // here we fill the initial array with random double numbers
        for (unsigned int i = 0; i < nr_rows_A * nr_cols_A; i++) {
                h_A[i] = fill_rand();
        }

        if (print) {
                cout << "A = [" << endl;
                print_matrix(h_A, nr_rows_A, nr_cols_A);
                cout << "];" << endl;
        }

        // Allocate 2 arrays on GPU
        double *d_A, *d_C;
        hipMalloc( &d_A, nr_rows_A * nr_cols_A * sizeof(double) ); cudaCheckError();
        hipMalloc( &d_C, nr_rows_C * nr_cols_C * sizeof(double) ); cudaCheckError();
        // Transfer the array to GPU
        hipMemcpy(d_A, h_A, nr_rows_A * nr_cols_A  * sizeof(double), hipMemcpyHostToDevice);
        cudaCheckError();
        // Αρχικοποίηση του πίνακα h_C με μηδενικά
        for (int i = 0; i < nr_rows_C * nr_cols_C; i++) {
                h_C[i] = 0.0;
        }
        hipMemcpy(d_C, h_C, nr_rows_C * nr_cols_C * sizeof(double), hipMemcpyHostToDevice);
        cudaCheckError();

        // Κάθε block θα έχει διάσταση
        unsigned int BLOCK_SIZE_PER_DIM = TILE_WIDTH;
        // Στρογγυλοποίηση προς τα πάνω για το πλήθος των block σε κάθε διάσταση
        unsigned int numBlocks = (imax(nr_rows_A,nr_cols_A)  - 1) / BLOCK_SIZE_PER_DIM + 1;
        // Ορισμός διαστάσεων πλέγματος
        dim3 dimGrid(numBlocks, numBlocks, 1);
        // Ορισμός διαστάσεων block
        dim3 dimBlock(BLOCK_SIZE_PER_DIM, BLOCK_SIZE_PER_DIM, 1);
        // cout << "\%dimGrid(" << numBlocks<< "," << numBlocks << ", 1) , dimBlock(";
        // cout << BLOCK_SIZE_PER_DIM << "," << BLOCK_SIZE_PER_DIM << ", 1);" << endl;
        hipEventCreate( &start); cudaCheckError();
        hipEventCreate( &stop); cudaCheckError();

        float totalTime[loops];
        int blockN = (nr_cols_A * nr_cols_A) / 1024 + 1;
        // Multiply A' and A on GPU
        for (int k = 0; k < loops; k++) {
                hipEventRecord(start); cudaCheckError();
                // Κλήση υπολογιστικού πυρήνα
                if (nr_rows_A==nr_cols_A) {
                        multiply<<<dimGrid, dimBlock>>>( d_A, nr_rows_A, nr_cols_A, d_C );
                }
                else {
                        dim3 dimGrid(nr_rows_C, nr_cols_C, 1);
                        multiplication<<<blockN,1024>>>(d_C, d_A, nr_cols_A, nr_rows_A);
                }
                cudaCheckError();
                hipDeviceSynchronize(); cudaCheckError();
                hipEventRecord(stop); cudaCheckError();

                // Copy the result on host memory
                hipMemcpy(h_C, d_C, nr_rows_C * nr_cols_C * sizeof(double), hipMemcpyDeviceToHost);
                cudaCheckError();
                hipEventSynchronize(stop); cudaCheckError();
                float milliseconds = 0;
                hipEventElapsedTime( &milliseconds, start, stop); cudaCheckError();
                cout << milliseconds << endl;
                totalTime[k] = milliseconds;
                // cout << "\%ex3, Dimensions :" << nr_rows_A<< "x" << nr_cols_C << " , Time (msec) : " << milliseconds << endl;
        }

        hipEventDestroy(start); cudaCheckError();
        hipEventDestroy(stop); cudaCheckError();

        float mo = 0.0;
        for(int k = 0; k < loops; k++) {
                mo += totalTime[k];
        }
        mo /= (loops);
        cout << endl << "average time = " << mo << endl;

        //Free GPU memory
        hipFree(d_A); cudaCheckError();
        hipFree(d_C); cudaCheckError();

        if (print) {
                cout << endl << "C = [" << endl;
                print_matrix(h_C, nr_rows_C, nr_cols_C);
                cout << "];" << endl;
                //Για επαλήθευση σε MATLAB
                cout<< "B=A'*A; D=B-C;" << endl;
        }

        // Free CPU memory
        free(h_A);
        free(h_C);

        return 0;
}
